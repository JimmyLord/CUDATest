#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include <assert.h>
#include "Utility/Helpers.h"

// CUDA Kernel function to add the elements of two arrays on the GPU
__global__
void addNElementsGPU1Thread(int n, float* x, float* y)
{
    for( int i = 0; i < n; i++ )
    {
        y[i] = x[i] + y[i];
    }
}

__global__
void addNElementsGPU1Block(int n, float* x, float* y)
{
    int index = threadIdx.x;
    int stride = blockDim.x;
    for( int i = index; i < n; i += stride)
    {
        y[i] = x[i] + y[i];
    }
}

__global__
void addNElementsGPUManyBlocks(int n, float* x, float* y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for( int i = index; i < n; i += stride)
    {
        y[i] = x[i] + y[i];
    }
}

double TaskAddValuesGPU(int numElements, int method)
{
    float* x;
    float* y;

    // Allocate Unified Memory � accessible from CPU or GPU.
    hipMallocManaged( &x, numElements*sizeof(float) );
    hipMallocManaged( &y, numElements*sizeof(float) );

    // Initialize x and y arrays on the host.
    for( int i=0; i<numElements; i++ )
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    double startTime = MyGetSystemTime();

    // Run kernel on all elements on the GPU.
    if( method == 0 )
    {
        addNElementsGPU1Thread<<<1, 1>>>( numElements, x, y );
    }
    if( method == 1 )
    {
        int blockSize = 256;
        addNElementsGPU1Block<<<1, blockSize>>>( numElements, x, y );
    }
    if( method == 2 )
    {
        int blockSize = 256;
        int numBlocks = (numElements + blockSize - 1) / blockSize;
        addNElementsGPUManyBlocks<<<numBlocks, blockSize>>>( numElements, x, y );
    }

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    double endTime = MyGetSystemTime();
    double runTime = endTime - startTime;

    // Check for errors (all values should be 3.0f).
    float maxError = 0.0f;
    for( int i=0; i<numElements; i++ )
    {
        float error = fabs( y[i] - 3.0f );
        maxError = fmax( maxError, error );
    }

    // There should be no error with this function.
    assert( maxError == 0.0f );

    // Free memory.
    hipFree( x );
    hipFree( y );

    return runTime;
}
